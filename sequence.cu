// Elapsed Real Time for input-4.txt: 0m1.124s

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of integers.
int *vList;

// Number of integers on the list.
int vCount = 0;

// Capacity of the list of integers.
int vCap = 0;

// Target sum.
int target_sum;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: sequence <target_sum> [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d\n", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}


//Kernel function to find subsequences that sum to the target
__global__ void checkSum(int *d_vList, int *d_results, int vCount_d, int target_sum, bool report) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (idx < vCount_d) {
    int local_count = 0;

    // Iterate through all subsequences starting at idx.
    int sequenceCount = 0;
    for (int k = idx; k >= 0; k--) {
      sequenceCount += d_vList[k];


      // Check if the local sum matches the target.
      if (sequenceCount == target_sum) {
        local_count++;
        if (report) {
          printf("I’m thread %d. Local count: %d. Sequence found at: %d-%d.\n", idx, local_count, k, idx);
        }
      }
    }

    if (report && local_count == 0){
      printf("I’m thread %d. Local count: 0.\n", idx);
    }
    // Store the local count in the results array.
    d_results[idx] = local_count;
  }
}

int main( int argc, char *argv[] ) {
  if ( argc < 2 || argc > 3 )
    usage();

  if ( sscanf( argv[ 1 ], "%d", &target_sum ) != 1)
    usage();

  bool report = false;
  if ( argc == 3 ) {
    if ( strcmp( argv[ 2 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();



  // Implementation start....

  // Add code to allocate memory on the device and copy over the list.
  int *d_vList, *d_results;
  hipMalloc((void **)&d_vList, vCount * sizeof(int));

  // Add code to copy the list over to the device.
  hipMalloc((void **)&d_results, vCount * sizeof(int));

  // Add code to allocate space on the device to hold the results.
  hipMemcpy(d_vList, vList, vCount * sizeof(int), hipMemcpyHostToDevice);

  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = (vCount + threadsPerBlock - 1) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions (you'll need to add some parameters)
  checkSum<<<blocksPerGrid, threadsPerBlock>>>(d_vList, d_results, vCount, target_sum, report);
  if (hipGetLastError() != hipSuccess)
    fail("Failure in CUDA kernel execution.");

  // Add code to copy results back to the host, add up all the per-thread counts 
  // and report one global total count.
  int *h_results = (int *) malloc(vCount * sizeof(int));
  hipMemcpy(h_results, d_results, vCount * sizeof(int), hipMemcpyDeviceToHost);
  int total_count = 0;
  for (int i = 0; i < vCount; i++) {
    total_count += h_results[i];
  }
  printf("Total count: %d\n", total_count);

  // Free memory on the device and the host.
  hipFree(d_vList);
  hipFree(d_results);
  free(vList);
  free(h_results);

  hipDeviceReset();
  return 0;
}
